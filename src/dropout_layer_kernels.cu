#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>

#include "dropout_layer.h"
#include "dark_cuda.h"
#include "utils.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id < size) input[id] = (rand[id] < prob) ? 0 : input[id]*scale;
}

void forward_dropout_layer_gpu(dropout_layer layer, network_state state)
{
    if (!state.train) return;
    int iteration_num = (*state.net.seen) / (state.net.batch*state.net.subdivisions);
    //if (iteration_num < state.net.burn_in) return;


    int size = layer.inputs*layer.batch;
    cuda_random(layer.rand_gpu, size);
    /*
    int i;
    for(i = 0; i < size; ++i){
        layer.rand[i] = rand_uniform();
    }
    cuda_push_array(layer.rand_gpu, layer.rand, size);
    */

    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >>>(state.input, size, layer.rand_gpu, layer.probability, layer.scale);
    CHECK_CUDA(hipPeekAtLastError());
}

void backward_dropout_layer_gpu(dropout_layer layer, network_state state)
{
    if(!state.delta) return;
    int iteration_num = (*state.net.seen) / (state.net.batch*state.net.subdivisions);
    //if (iteration_num < state.net.burn_in) return;

    int size = layer.inputs*layer.batch;

    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >>>(state.delta, size, layer.rand_gpu, layer.probability, layer.scale);
    CHECK_CUDA(hipPeekAtLastError());
}
