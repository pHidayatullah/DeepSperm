#include "hip/hip_runtime.h"
#include "darknet.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>

#include "activations.h"
#include "dark_cuda.h"


__device__ float lhtan_activate_kernel(float x)
{
    if(x < 0) return .001*x;
    if(x > 1) return .001*(x-1) + 1;
    return x;
}
__device__ float lhtan_gradient_kernel(float x)
{
    if(x > 0 && x < 1) return 1;
    return .001;
}

__device__ float hardtan_activate_kernel(float x)
{
    if (x < -1) return -1;
    if (x > 1) return 1;
    return x;
}
__device__ float linear_activate_kernel(float x){return x;}
__device__ float logistic_activate_kernel(float x){return 1.f/(1.f + expf(-x));}
__device__ float loggy_activate_kernel(float x){return 2.f/(1.f + expf(-x)) - 1;}
__device__ float relu_activate_kernel(float x){return x*(x>0);}
__device__ float elu_activate_kernel(float x){return (x >= 0)*x + (x < 0)*(expf(x)-1);}
__device__ float selu_activate_kernel(float x) { return (x >= 0)*1.0507f*x + (x < 0)*1.0507f*1.6732f*(expf(x) - 1); }
__device__ float relie_activate_kernel(float x){return (x>0) ? x : .01f*x;}
__device__ float ramp_activate_kernel(float x){return x*(x>0)+.1f*x;}
__device__ float leaky_activate_kernel(float x){return (x>0) ? x : .1f*x;}
__device__ float tanh_activate_kernel(float x){return (2/(1 + expf(-2*x)) - 1);}
__device__ float plse_activate_kernel(float x)
{
    if(x < -4) return .01f * (x + 4);
    if(x > 4)  return .01f * (x - 4) + 1;
    return .125f*x + .5f;
}
__device__ float stair_activate_kernel(float x)
{
    int n = floorf(x);
    if (n%2 == 0) return floorf(x/2.f);
    else return (x - n) + floorf(x/2.f);
}


__device__ float hardtan_gradient_kernel(float x)
{
    if (x > -1 && x < 1) return 1;
    return 0;
}
__device__ float linear_gradient_kernel(float x){return 1;}
__device__ float logistic_gradient_kernel(float x){return (1-x)*x;}
__device__ float loggy_gradient_kernel(float x)
{
    float y = (x+1.F)/2.F;
    return 2*(1-y)*y;
}
__device__ float relu_gradient_kernel(float x){return (x>0);}
__device__ float elu_gradient_kernel(float x){return (x >= 0) + (x < 0)*(x + 1);}
__device__ float selu_gradient_kernel(float x) { return (x >= 0)*1.0507f + (x < 0)*(x + 1.0507f*1.6732f); }
__device__ float relie_gradient_kernel(float x){return (x>0) ? 1 : .01f;}
__device__ float ramp_gradient_kernel(float x){return (x>0)+.1f;}
__device__ float leaky_gradient_kernel(float x){return (x>0) ? 1 : .1f;}
__device__ float tanh_gradient_kernel(float x){return 1-x*x;}
__device__ float plse_gradient_kernel(float x){return (x < 0 || x > 1) ? .01f : .125f;}
__device__ float stair_gradient_kernel(float x)
{
    if (floor(x) == x) return 0;
    return 1;
}

__device__ float activate_kernel(float x, ACTIVATION a)
{
    switch(a){
        case LINEAR:
            return linear_activate_kernel(x);
        case LOGISTIC:
            return logistic_activate_kernel(x);
        case LOGGY:
            return loggy_activate_kernel(x);
        case RELU:
            return relu_activate_kernel(x);
        case ELU:
            return elu_activate_kernel(x);
        case SELU:
            return selu_activate_kernel(x);
        case RELIE:
            return relie_activate_kernel(x);
        case RAMP:
            return ramp_activate_kernel(x);
        case LEAKY:
            return leaky_activate_kernel(x);
        case TANH:
            return tanh_activate_kernel(x);
        case PLSE:
            return plse_activate_kernel(x);
        case STAIR:
            return stair_activate_kernel(x);
        case HARDTAN:
            return hardtan_activate_kernel(x);
        case LHTAN:
            return lhtan_activate_kernel(x);
    }
    return 0;
}

__device__ float gradient_kernel(float x, ACTIVATION a)
{
    switch (a) {
    case LINEAR:
        return linear_gradient_kernel(x);
    case LOGISTIC:
        return logistic_gradient_kernel(x);
    case LOGGY:
        return loggy_gradient_kernel(x);
    case RELU:
        return relu_gradient_kernel(x);
    case ELU:
        return elu_gradient_kernel(x);
    case SELU:
        return selu_gradient_kernel(x);
    case RELIE:
        return relie_gradient_kernel(x);
    case RAMP:
        return ramp_gradient_kernel(x);
    case LEAKY:
        return leaky_gradient_kernel(x);
    case TANH:
        return tanh_gradient_kernel(x);
    case PLSE:
        return plse_gradient_kernel(x);
    case STAIR:
        return stair_gradient_kernel(x);
    case HARDTAN:
        return hardtan_gradient_kernel(x);
    case LHTAN:
        return lhtan_gradient_kernel(x);
    }
    return 0;
}

__global__ void binary_gradient_array_kernel(float *x, float *dy, int n, int s, BINARY_ACTIVATION a, float *dx)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int i = id % s;
    int b = id / s;
    float x1 = x[b*s + i];
    float x2 = x[b*s + s / 2 + i];
    if (id < n) {
        float de = dy[id];
        dx[b*s + i] = x2*de;
        dx[b*s + s / 2 + i] = x1*de;
    }
}

extern "C" void binary_gradient_array_gpu(float *x, float *dx, int n, int size, BINARY_ACTIVATION a, float *y)
{
    binary_gradient_array_kernel << <cuda_gridsize(n / 2), BLOCK, 0, get_cuda_stream() >> >(x, dx, n / 2, size, a, y);
    CHECK_CUDA(hipPeekAtLastError());
}
__global__ void binary_activate_array_kernel(float *x, int n, int s, BINARY_ACTIVATION a, float *y)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int i = id % s;
    int b = id / s;
    float x1 = x[b*s + i];
    float x2 = x[b*s + s / 2 + i];
    if (id < n) y[id] = x1*x2;
}

extern "C" void binary_activate_array_gpu(float *x, int n, int size, BINARY_ACTIVATION a, float *y)
{
    binary_activate_array_kernel << <cuda_gridsize(n / 2), BLOCK, 0, get_cuda_stream() >> >(x, n / 2, size, a, y);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void activate_array_kernel(float *x, int n, ACTIVATION a)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n) x[i] = activate_kernel(x[i], a);
}



__global__ void activate_array_swish_kernel(float *x, int n, float *output_sigmoid_gpu, float *output_gpu)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        float x_val = x[i];
        float sigmoid = logistic_activate_kernel(x_val);
        output_sigmoid_gpu[i] = sigmoid;
        output_gpu[i] = x_val * sigmoid;
    }
}

__global__ void activate_array_leaky_kernel(float *x, int n)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        x[index] = leaky_activate_kernel(x[index]);
    }
}

__global__ void activate_array_selu_kernel(float *x, int n)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        x[index] = selu_activate_kernel(x[index]);
    }
}

__global__ void activate_array_logistic_kernel(float *x, int n)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        x[index] = logistic_activate_kernel(x[index]);
    }
}

__global__ void activate_array_tanh_kernel(float *x, int n)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        x[index] = tanh_activate_kernel(x[index]);
    }
}

__global__ void activate_array_hardtan_kernel(float *x, int n)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        x[index] = hardtan_activate_kernel(x[index]);
    }
}

__global__ void activate_array_relu_kernel(float *x, int n)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        x[index] = relu_activate_kernel(x[index]);
    }
}

__global__ void gradient_array_kernel(float *x, int n, ACTIVATION a, float *delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n) delta[i] *= gradient_kernel(x[i], a);
}

// https://github.com/BVLC/caffe/blob/04ab089db018a292ae48d51732dd6c66766b36b6/src/caffe/layers/swish_layer.cu#L28-L30
__global__ void gradient_array_swish_kernel(float *x, int n, float *sigmoid_gpu, float *delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        float swish = x[i];
        delta[i] *= swish + sigmoid_gpu[i] * (1 - swish); // gradient_kernel(x[i], a);
    }
}

__global__ void gradient_array_leaky_kernel(float *x, int n, float *delta)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        delta[index] *= leaky_gradient_kernel(x[index]);
    }
}

__global__ void gradient_array_selu_kernel(float *x, int n, float *delta)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        delta[index] *= selu_gradient_kernel(x[index]);
    }
}

__global__ void gradient_array_logistic_kernel(float *x, int n, float *delta)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        delta[index] *= logistic_gradient_kernel(x[index]);
    }
}

__global__ void gradient_array_tanh_kernel(float *x, int n, float *delta)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        delta[index] *= tanh_gradient_kernel(x[index]);
    }
}

__global__ void gradient_array_hardtan_kernel(float *x, int n, float *delta)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        delta[index] *= hardtan_gradient_kernel(x[index]);
    }
}

__global__ void gradient_array_relu_kernel(float *x, int n, float *delta)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        delta[index] *= relu_gradient_kernel(x[index]);
    }
}

extern "C" void activate_array_ongpu(float *x, int n, ACTIVATION a)
{
    const int num_blocks = get_number_of_blocks(n, BLOCK);
    if (a == LINEAR) return;
    else if(a == LEAKY) activate_array_leaky_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n);
    else if (a == LOGISTIC) activate_array_logistic_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n);
    else if (a == TANH) activate_array_tanh_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n);
    else if (a == HARDTAN) activate_array_hardtan_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n);
    else if (a == RELU) activate_array_relu_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n);
    else if (a == SELU) activate_array_selu_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n);
    else
        activate_array_kernel<<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream()>>>(x, n, a);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void activate_array_swish_ongpu(float *x, int n, float *output_sigmoid_gpu, float *output_gpu)
{
    const int num_blocks = get_number_of_blocks(n, BLOCK);
    activate_array_swish_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> >(x, n, output_sigmoid_gpu, output_gpu);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void gradient_array_ongpu(float *x, int n, ACTIVATION a, float *delta)
{
    const int num_blocks = get_number_of_blocks(n, BLOCK);
    if (a == LINEAR) return;
    else if (a == LEAKY) gradient_array_leaky_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n, delta);
    else if (a == LOGISTIC) gradient_array_logistic_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n, delta);
    else if (a == TANH) gradient_array_tanh_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n, delta);
    else if (a == HARDTAN) gradient_array_hardtan_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n, delta);
    else if (a == RELU) gradient_array_relu_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n, delta);
    else if (a == SELU) gradient_array_selu_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n, delta);
    else
        gradient_array_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> > (x, n, a, delta);
    CHECK_CUDA(hipPeekAtLastError());
}


extern "C" void gradient_array_swish_ongpu(float *x, int n, float *sigmoid_gpu, float *delta)
{
    const int num_blocks = get_number_of_blocks(n, BLOCK);
    gradient_array_swish_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> > (x, n, sigmoid_gpu, delta);
    CHECK_CUDA(hipPeekAtLastError());
}