#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <assert.h>

#include "blas.h"
#include "dark_cuda.h"
#include "utils.h"
#include "tree.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if(offset < size) output[(batch*n+filter)*size + offset] *= biases[filter];
}

void scale_bias_gpu(float *output, float *biases, int batch, int n, int size)
{
    dim3 dimGrid((size-1)/BLOCK + 1, n, batch);
    dim3 dimBlock(BLOCK, 1, 1);

    scale_bias_kernel<<<dimGrid, dimBlock, 0, get_cuda_stream()>>>(output, biases, n, size);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int i,b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;
    for(b = 0; b < batch; ++b){
        for(i = 0; i < size; i += BLOCK){
            int index = p + i + size*(filter + n*b);
            sum += (p+i < size) ? delta[index]*x_norm[index] : 0;
        }
    }
    part[p] = sum;
    __syncthreads();
    if (p == 0) {
        for(i = 0; i < BLOCK; ++i) scale_updates[filter] += part[i];
    }
}

void backward_scale_gpu(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    backward_scale_kernel<<<n, BLOCK, 0, get_cuda_stream() >>>(x_norm, delta, batch, n, size, scale_updates);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if(offset < size) output[(batch*n+filter)*size + offset] += biases[filter];
}

void add_bias_gpu(float *output, float *biases, int batch, int n, int size)
{
    dim3 dimGrid((size-1)/BLOCK + 1, n, batch);
    dim3 dimBlock(BLOCK, 1, 1);

    add_bias_kernel<<<dimGrid, dimBlock, 0, get_cuda_stream()>>>(output, biases, n, size);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int i,b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;
    for(b = 0; b < batch; ++b){
        for(i = 0; i < size; i += BLOCK){
            int index = p + i + size*(filter + n*b);
            sum += (p+i < size) ? delta[index] : 0;
        }
    }
    part[p] = sum;
    __syncthreads();
    if (p == 0) {
        for(i = 0; i < BLOCK; ++i) bias_updates[filter] += part[i];
    }
}

/*
__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;
    int b, i;
    for(b = 0; b <  batch; ++b){
        for(i = 0; i < size; ++i){
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            sum += output[i1] * output[i2];
            norm1 += output[i1] * output[i1];
            norm2 += output[i2] * output[i2];
        }
    }
    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    sum = sum / norm;
    for(b = 0; b <  batch; ++b){
        for(i = 0; i < size; ++i){
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            delta[i1] += - scale * sum * output[i2] / norm;
            delta[i2] += - scale * sum * output[i1] / norm;
        }
    }
}

void dot_error_gpu(layer l)
{
    dot_kernel<<<cuda_gridsize(l.n*l.n), BLOCK, 0, get_cuda_stream()>>>(l.output_gpu, l.dot, l.batch, l.n, l.out_w * l.out_h, l.delta_gpu);
    CHECK_CUDA(hipPeekAtLastError());
}
*/

void backward_bias_gpu(float *bias_updates, float *delta, int batch, int n, int size)
{
    backward_bias_kernel<<<n, BLOCK, 0, get_cuda_stream() >>>(bias_updates, delta, batch, n, size);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;

    float mhat = m[index] / (1.f - powf(B1, t));
    float vhat = v[index] / (1.f - powf(B2, t));

    x[index] = x[index] + rate * mhat / (sqrtf(vhat) + eps);
}

extern "C" void adam_gpu(int n, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    adam_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> >(n, x, m, v, B1, B2, rate, eps, t);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void adam_update_gpu(float *w, float *d, float *m, float *v, float B1, float B2, float eps, float decay, float rate, int n, int batch, int t)
{
    scal_ongpu(n, B1, m, 1);
    scal_ongpu(n, B2, v, 1);
    axpy_ongpu(n, -decay*batch, w, 1, d, 1);

    axpy_ongpu(n, (1 - B1), d, 1, m, 1);
    mul_ongpu(n, d, 1, d, 1);
    axpy_ongpu(n, (1 - B2), d, 1, v, 1);

    adam_gpu(n, w, m, v, B1, B2, rate, eps, t);
    fill_ongpu(n, 0, d, 1);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;

    x[index] = (x[index] - mean[f])/(sqrtf(variance[f]) + .000001f);
}

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;

    delta[index] = delta[index] * 1.F/(sqrtf(variance[f]) + .000001f) + variance_delta[f] * 2. * (x[index] - mean[f]) / (spatial * batch) + mean_delta[f]/(spatial*batch);
}

extern "C" void normalize_delta_gpu(float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    size_t N = batch*filters*spatial;
    normalize_delta_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >>>(N, x, mean, variance, mean_delta, variance_delta, batch, filters, spatial, delta);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void  variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    int j,k;
    variance_delta[i] = 0;
    for(j = 0; j < batch; ++j){
        for(k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            variance_delta[i] += delta[index]*(x[index] - mean[i]);
        }
    }
    variance_delta[i] *= -.5 * powf(variance[i] + .000001f, (float)(-3./2.));
}

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int k;
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;
    sum[i] = 0;
    for(k = 0; k < n; ++k){
        sum[i] += x[k*groups + i];
    }
}

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;
            local[id] += (i+id < spatial) ? delta[index] : 0;
        }
    }
    __syncthreads();

    if(id == 0){
        mean_delta[filter] = 0;
        for(i = 0; i < threads; ++i){
            mean_delta[filter] += local[i];
        }
        mean_delta[filter] *= (-1.F/sqrtf(variance[filter] + .000001f));
    }
}

__global__ void  fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;

            local[id] += (i+id < spatial) ? delta[index]*(x[index] - mean[filter]) : 0;
        }
    }
    __syncthreads();

    if(id == 0){
        variance_delta[filter] = 0;
        for(i = 0; i < threads; ++i){
            variance_delta[filter] += local[i];
        }
        variance_delta[filter] *= -.5 * powf(variance[filter] + .000001f, (float)(-3./2.));
    }
}


__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    int j,k;
    mean_delta[i] = 0;
    for (j = 0; j < batch; ++j) {
        for (k = 0; k < spatial; ++k) {
            int index = j*filters*spatial + i*spatial + k;
            mean_delta[i] += delta[index];
        }
    }
    mean_delta[i] *= (-1.F/sqrtf(variance[i] + .000001f));
}

extern "C" void mean_delta_gpu(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    mean_delta_kernel<<<cuda_gridsize(filters), BLOCK, 0, get_cuda_stream() >>>(delta, variance, batch, filters, spatial, mean_delta);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void fast_mean_delta_gpu(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    fast_mean_delta_kernel<<<filters, BLOCK, 0, get_cuda_stream() >>>(delta, variance, batch, filters, spatial, mean_delta);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void fast_variance_delta_gpu(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    fast_variance_delta_kernel<<<filters, BLOCK, 0, get_cuda_stream() >>>(x, delta, mean, variance, batch, filters, spatial, variance_delta);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void  mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.F/(batch * spatial);
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    int j,k;
    mean[i] = 0;
    for(j = 0; j < batch; ++j){
        for(k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            mean[i] += x[index];
        }
    }
    mean[i] *= scale;
}

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1.F/(batch * spatial - 1);
    int j,k;
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    variance[i] = 0;
    for(j = 0; j < batch; ++j){
        for(k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            variance[i] += powf((x[index] - mean[i]), 2);
        }
    }
    variance[i] *= scale;
}

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i >= N) return;
    int in_index = i;
    int in_w = i%w;
    i = i/w;
    int in_h = i%h;
    i = i/h;
    int in_c = i%c;
    i = i/c;
    int b = i%batch;

    int out_c = c/(stride*stride);

    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w*stride + offset % stride;
    int h2 = in_h*stride + offset / stride;
    //printf("%d\n", offset);
    int out_index = w2 + w*stride*(h2 + h*stride*(c2 + out_c*b));

   // printf("%d %d %d\n", w2, h2, c2);
    //printf("%d %d\n", in_index, out_index);
    //if(out_index >= N || out_index < 0) printf("bad bad bad \n");

    if(forward) out[out_index] = x[in_index];
    else out[in_index] = x[out_index];
    //if(forward) out[1] = x[1];
    //else out[0] = x[0];
}

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX,  float *Y, int OFFY, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[OFFY+i*INCY] += ALPHA*X[OFFX+i*INCX];
}

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY] = powf(X[i*INCX], ALPHA);
}

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = ALPHA;
}

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i*INCX]));
}

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
        if((X[i*INCX] * X[i*INCX]) < (ALPHA * ALPHA)) X[i*INCX] = 0;
    }
}

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] *= ALPHA;
}

__global__ void scal_add_kernel(int N, float ALPHA, float BETA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) X[i*INCX] = X[i*INCX] * ALPHA + BETA;
}

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = ALPHA;
}

__global__ void mask_kernel_new_api(int n, float *x, float mask_num, float *mask, float val)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n && mask[i] == mask_num) x[i] = val;
}

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n && mask[i] == mask_num) x[i] = mask_num;
}

__global__ void copy_kernel(int N,  float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY + OFFY] = X[i*INCX + OFFX];
}

__global__ void simple_copy_kernel(int size, float *src, float *dst)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size)
        dst[index] = src[index];
}

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY] *= X[i*INCX];
}


extern "C" void normalize_gpu(float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    size_t N = batch*filters*spatial;
    normalize_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream()>>>(N, x, mean, variance, batch, filters, spatial);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void  fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;
            local[id] += (i+id < spatial) ? x[index] : 0;
        }
    }
    __syncthreads();

    if(id == 0){
        mean[filter] = 0;
        for(i = 0; i < threads; ++i){
            mean[filter] += local[i];
        }
        mean[filter] /= spatial * batch;
    }
}

__global__ void  fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;

            local[id] += (i+id < spatial) ? powf((x[index] - mean[filter]), 2) : 0;
        }
    }
    __syncthreads();

    if(id == 0){
        variance[filter] = 0;
        for(i = 0; i < threads; ++i){
            variance[filter] += local[i];
        }
        variance[filter] /= (spatial * batch - 1);
    }
}

extern "C" void fast_mean_gpu(float *x, int batch, int filters, int spatial, float *mean)
{
    fast_mean_kernel<<<filters, BLOCK, 0, get_cuda_stream()>>>(x, batch, filters, spatial, mean);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void fast_variance_gpu(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    fast_variance_kernel<<<filters, BLOCK, 0, get_cuda_stream() >>>(x, mean, batch, filters, spatial, variance);
    CHECK_CUDA(hipPeekAtLastError());
}


extern "C" void mean_gpu(float *x, int batch, int filters, int spatial, float *mean)
{
    mean_kernel<<<cuda_gridsize(filters), BLOCK, 0, get_cuda_stream() >>>(x, batch, filters, spatial, mean);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void variance_gpu(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    variance_kernel<<<cuda_gridsize(filters), BLOCK, 0, get_cuda_stream() >>>(x, mean, batch, filters, spatial, variance);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void axpy_ongpu(int N, float ALPHA, float * X, int INCX, float * Y, int INCY)
{
    axpy_ongpu_offset(N, ALPHA, X, 0, INCX, Y, 0, INCY);
}

extern "C" void pow_ongpu(int N, float ALPHA, float * X, int INCX, float * Y, int INCY)
{
    pow_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >>>(N, ALPHA, X, INCX, Y, INCY);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void axpy_ongpu_offset(int N, float ALPHA, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY)
{
    axpy_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream()>>>(N, ALPHA, X, OFFX, INCX, Y, OFFY, INCY);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void copy_ongpu(int N, float * X, int INCX, float * Y, int INCY)
{
    copy_ongpu_offset(N, X, 0, INCX, Y, 0, INCY);
}

extern "C" void simple_copy_ongpu(int size, float *src, float *dst)
{
    const int num_blocks = size / BLOCK + 1;
    simple_copy_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(size, src, dst);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void mul_ongpu(int N, float * X, int INCX, float * Y, int INCY)
{
    mul_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >>>(N, X, INCX, Y, INCY);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void copy_ongpu_offset(int N, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY)
{
    copy_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream()>>>(N, X, OFFX, INCX, Y, OFFY, INCY);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i >= N) return;
    int in_s = i%spatial;
    i = i/spatial;
    int in_c = i%layers;
    i = i/layers;
    int b = i;

    int i1 = b*layers*spatial + in_c*spatial + in_s;
    int i2 = b*layers*spatial + in_s*layers +  in_c;

    if (forward) out[i2] = x[i1];
    else out[i1] = x[i2];
}

extern "C" void flatten_ongpu(float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int size = spatial*batch*layers;
    flatten_kernel<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream()>>>(size, x, spatial, layers, batch, forward, out);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void reorg_ongpu(float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int size = w*h*c*batch;
    reorg_kernel<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream()>>>(size, x, w, h, c, batch, stride, forward, out);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void mask_gpu_new_api(int N, float * X, float mask_num, float * mask, float val)
{
	mask_kernel_new_api <<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >>>(N, X, mask_num, mask, val);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void mask_ongpu(int N, float * X, float mask_num, float * mask)
{
    mask_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >>>(N, X, mask_num, mask);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void const_ongpu(int N, float ALPHA, float * X, int INCX)
{
    const_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >>>(N, ALPHA, X, INCX);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void constrain_ongpu(int N, float ALPHA, float * X, int INCX)
{
    constrain_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >>>(N, ALPHA, X, INCX);
    CHECK_CUDA(hipPeekAtLastError());
}


extern "C" void scal_ongpu(int N, float ALPHA, float * X, int INCX)
{
    scal_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream()>>>(N, ALPHA, X, INCX);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void scal_add_ongpu(int N, float ALPHA, float BETA, float * X, int INCX)
{
    scal_add_kernel << <cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >> >(N, ALPHA, BETA, X, INCX);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void supp_ongpu(int N, float ALPHA, float * X, int INCX)
{
    supp_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >>>(N, ALPHA, X, INCX);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void fill_ongpu(int N, float ALPHA, float * X, int INCX)
{
    fill_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream()>>>(N, ALPHA, X, INCX);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int i = id % minw;
    id /= minw;
    int j = id % minh;
    id /= minh;
    int k = id % minc;
    id /= minc;
    int b = id % batch;

    int out_index = i*sample + w2*(j*sample + h2*(k + c2*b));
    int add_index = i*stride + w1*(j*stride + h1*(k + c1*b));
    out[out_index] += add[add_index];
}

extern "C" void shortcut_gpu(int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int minw = (w1 < w2) ? w1 : w2;
    int minh = (h1 < h2) ? h1 : h2;
    int minc = (c1 < c2) ? c1 : c2;

    int stride = w1/w2;
    int sample = w2/w1;
    assert(stride == h1/h2);
    assert(sample == h2/h1);
    if(stride < 1) stride = 1;
    if(sample < 1) sample = 1;

    int size = batch * minw * minh * minc;
    shortcut_kernel<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream()>>>(size, minw, minh, minc, stride, sample, batch, w1, h1, c1, add, w2, h2, c2, out);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void simple_input_shortcut_kernel(float *in, int size, float *add, float *out)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;

    out[id] = in[id] + add[id];
}

__global__ void input_shortcut_kernel(float *in, int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int i = id % minw;
    id /= minw;
    int j = id % minh;
    id /= minh;
    int k = id % minc;
    id /= minc;
    int b = id % batch;

    int out_index = i*sample + w2*(j*sample + h2*(k + c2*b));
    int add_index = i*stride + w1*(j*stride + h1*(k + c1*b));
    out[out_index] = in[out_index] + add[add_index];
}

extern "C" void input_shortcut_gpu(float *in, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    if (w1 == w2 && h1 == h2 && c1 == c2) {
        int size = batch * w1 * h1 * c1;
        simple_input_shortcut_kernel << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> >(in, size, add, out);
        CHECK_CUDA(hipPeekAtLastError());
        return;
    }

    int minw = (w1 < w2) ? w1 : w2;
    int minh = (h1 < h2) ? h1 : h2;
    int minc = (c1 < c2) ? c1 : c2;

    int stride = w1 / w2;
    int sample = w2 / w1;
    assert(stride == h1 / h2);
    assert(sample == h2 / h1);
    if (stride < 1) stride = 1;
    if (sample < 1) sample = 1;

    int size = batch * minw * minh * minc;
    //input_shortcut_kernel << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> >(in, size, minw, minh, minc, stride, sample, batch, w1, h1, c1, add, w2, h2, c2, out);
    simple_copy_ongpu(w2 * h2 * c2 * batch, in, out);
    shortcut_kernel << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> >(size, minw, minh, minc, stride, sample, batch, w1, h1, c1, add, w2, h2, c2, out);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        float diff = truth[i] - pred[i];
        float abs_val = abs(diff);
        if(abs_val < 1) {
            error[i] = diff * diff;
            delta[i] = diff;
        }
        else {
            error[i] = 2*abs_val - 1;
            delta[i] = (diff < 0) ? -1 : 1;
        }
    }
}

extern "C" void smooth_l1_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
    smooth_l1_kernel<<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>>(n, pred, truth, delta, error);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		float t = truth[i];
		float p = pred[i];
		error[i] = (t) ? -log(p) : 0;
		delta[i] = t - p;
	}
}

extern "C" void softmax_x_ent_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
	softmax_x_ent_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> >(n, pred, truth, delta, error);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        float diff = truth[i] - pred[i];
        error[i] = diff * diff; //I know this is technically wrong, deal with it.
        delta[i] = diff;
    }
}

extern "C" void l2_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
    l2_kernel<<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>>(n, pred, truth, delta, error);
    CHECK_CUDA(hipPeekAtLastError());
}



__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        c[i] = s[i]*a[i] + (1-s[i])*(b ? b[i] : 0);
    }
}

extern "C" void weighted_sum_gpu(float *a, float *b, float *s, int num, float *c)
{
    weighted_sum_kernel<<<cuda_gridsize(num), BLOCK, 0, get_cuda_stream() >>>(num, a, b, s, c);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        if(da) da[i] += dc[i] * s[i];
        db[i] += dc[i] * (1-s[i]);
        ds[i] += dc[i] * a[i] + dc[i] * -b[i];
    }
}

extern "C" void weighted_delta_gpu(float *a, float *b, float *s, float *da, float *db, float *ds, int num, float *dc)
{
    weighted_delta_kernel<<<cuda_gridsize(num), BLOCK, 0, get_cuda_stream() >>>(num, a, b, s, da, db, ds, dc);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        c[i] += a[i]*b[i];
    }
}

extern "C" void mult_add_into_gpu(int num, float *a, float *b, float *c)
{
    mult_add_into_kernel<<<cuda_gridsize(num), BLOCK, 0, get_cuda_stream() >>>(num, a, b, c);
    CHECK_CUDA(hipPeekAtLastError());
}


__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    for(i = 0; i < n; ++i){
        int val = input[i];
        largest = (val>largest) ? val : largest;
    }
    for(i = 0; i < n; ++i){
        float e = exp(input[i]/temp - largest/temp);
        sum += e;
        output[i] = e;
    }
    for(i = 0; i < n; ++i){
        output[i] /= sum;
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output)
{
    int b = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(b >= batch) return;
    softmax_device(n, input + b*offset, temp, output + b*offset);
}

extern "C" void softmax_gpu(float *input, int n, int offset, int groups, float temp, float *output)
{
    int inputs = n;
    int batch = groups;
    softmax_kernel<<<cuda_gridsize(batch), BLOCK, 0, get_cuda_stream()>>>(inputs, offset, batch, input, temp, output);
    CHECK_CUDA(hipPeekAtLastError());
}

__device__ void softmax_device_new_api(float *input, int n, float temp, int stride, float *output)
{
	int i;
	float sum = 0;
	float largest = -INFINITY;
	for (i = 0; i < n; ++i) {
		int val = input[i*stride];
		largest = (val>largest) ? val : largest;
	}
	for (i = 0; i < n; ++i) {
		float e = expf(input[i*stride] / temp - largest / temp);
		sum += e;
		output[i*stride] = e;
	}
	for (i = 0; i < n; ++i) {
		output[i*stride] /= sum;
	}
}

__global__ void softmax_kernel_new_api(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
	int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= batch*groups) return;
	int b = id / groups;
	int g = id % groups;
	softmax_device_new_api(input + b*batch_offset + g*group_offset, n, temp, stride, output + b*batch_offset + g*group_offset);
}

extern "C" void softmax_gpu_new_api(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
	softmax_kernel_new_api << <cuda_gridsize(batch*groups), BLOCK, 0, get_cuda_stream() >> >(input, n, batch, batch_offset, groups, group_offset, stride, temp, output);
    CHECK_CUDA(hipPeekAtLastError());
}


__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= N) return;
    int out_index = i;
    int out_w = i % (w*stride);
    i = i / (w*stride);
    int out_h = i % (h*stride);
    i = i / (h*stride);
    int out_c = i%c;
    i = i / c;
    int b = i%batch;

    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_c = out_c;

    int in_index = b*w*h*c + in_c*w*h + in_h*w + in_w;


    if (forward) out[out_index] += scale * x[in_index];
    else atomicAdd(x + in_index, scale * out[out_index]);
}

extern "C" void upsample_gpu(float *in, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t size = w*h*c*batch*stride*stride;
    upsample_kernel << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> >(size, in, w, h, c, batch, stride, forward, scale, out);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
	int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= spatial*batch*groups) return;
	int s = id % spatial;
	id = id / spatial;
	int g = id % groups;
	int b = id / groups;
	int goff = group_offset[g] * spatial;
	int boff = b*stride;
	softmax_device_new_api(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}

extern "C" void softmax_tree_gpu(float *input, int spatial, int batch, int stride, float temp, float *output, tree hier)
{
	int *tree_groups_size = cuda_make_int_array_new_api(hier.group_size, hier.groups);
	int *tree_groups_offset = cuda_make_int_array_new_api(hier.group_offset, hier.groups);
	/*
	static int *tree_groups_size = 0;
	static int *tree_groups_offset = 0;
	if(!tree_groups_size){
	tree_groups_size = cuda_make_int_array(hier.group_size, hier.groups);
	tree_groups_offset = cuda_make_int_array(hier.group_offset, hier.groups);
	}
	*/
	int num = spatial*batch*hier.groups;
	softmax_tree_kernel <<<cuda_gridsize(num), BLOCK, 0, get_cuda_stream() >>>(input, spatial, batch, stride, temp, output, hier.groups, tree_groups_size, tree_groups_offset);
    CHECK_CUDA(hipPeekAtLastError());
	cuda_free((float *)tree_groups_size);
	cuda_free((float *)tree_groups_offset);
}


__global__ void fix_nan_and_inf_kernel(float *input, size_t size)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size) {
        float val = input[index];
        if (isnan(val) || isinf(val))
            input[index] = 1.0f / index;  // pseudo random value
    }
}

extern "C" void fix_nan_and_inf(float *input, size_t size)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    fix_nan_and_inf_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> >(input, size);
    CHECK_CUDA(hipPeekAtLastError());
    //CHECK_CUDA(hipDeviceSynchronize());
}


__global__ void is_nan_or_inf_kernel(float *input, size_t size, int *pinned_return)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size) {
        float val = input[index];
        if (isnan(val) || isinf(val))
            *pinned_return = 1;
    }
}

extern "C" int is_nan_or_inf(float *input, size_t size)
{
    int *pinned_return;
    CHECK_CUDA(hipHostAlloc(&pinned_return, sizeof(int), hipHostRegisterMapped));
    *pinned_return = 0;

    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    is_nan_or_inf_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> >(input, size, pinned_return);
    CHECK_CUDA(hipDeviceSynchronize());
    int ret_val = *pinned_return;

    CHECK_CUDA(hipHostFree(pinned_return));
    return ret_val;
}

__global__ void add_3_arrays_activate_kernel(float *a1, float *a2, float *a3, size_t size, ACTIVATION a, float *dst)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size) {
        float val = 0;
        val += a1[index];
        val += a2[index];
        if (a3) val += a3[index];
        if (a == LOGISTIC) val = 1.f / (1.f + expf(-val));
        else if(a == TANH) val = (2 / (1 + expf(-2 * val)) - 1);
        dst[index] = val;
    }
}

extern "C" void add_3_arrays_activate(float *a1, float *a2, float *a3, size_t size, ACTIVATION a, float *dst)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    if (a != LOGISTIC && a != TANH) {
        printf(" add_3_arrays_activate() doesn't support activation %d, it supports only LOGISTIC and TANH \n", a);
        exit(EXIT_FAILURE);
    }
    add_3_arrays_activate_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> >(a1, a2, a3, size, a, dst);
}


__global__ void sum_of_mults_kernel(float *a1, float *a2, float *b1, float *b2, size_t size, float *dst)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size) {
        dst[index] = a1[index] * a2[index] + b1[index] * b2[index];
    }
}

extern "C" void sum_of_mults(float *a1, float *a2, float *b1, float *b2,  size_t size, float *dst)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    sum_of_mults_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> >(a1, a2, b1, b2, size, dst);
}


__global__ void activate_and_mult_kernel(float *a1, float *a2, size_t size, ACTIVATION a, float *dst)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size) {
        float val = a1[index];
        if (a == TANH) val = (2 / (1 + expf(-2 * val)) - 1);
        dst[index] = val * a2[index];
    }
}

extern "C" void activate_and_mult(float *a1, float *a2, size_t size, ACTIVATION a, float *dst)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    if (a != TANH) {
        printf(" activat_and_mult() doesn't support activation %d, it supports only TANH \n", a);
        exit(EXIT_FAILURE);
    }
    activate_and_mult_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> >(a1, a2, size, a, dst);
}



__global__ void scale_channels_kernel(float *in_w_h_c, int size, int channel_size, float *scales_c, float *out)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size) {
        out[index] = in_w_h_c[index] * scales_c[index / channel_size];
    }
}

extern "C" void scale_channels_gpu(float *in_w_h_c, int size, int channel_size, float *scales_c, float *out)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    scale_channels_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> >(in_w_h_c, size, channel_size, scales_c, out);
    CHECK_CUDA(hipPeekAtLastError());
}


__inline__ __device__
float warpAllReduceSum(float val) {
    for (int mask = WARP_SIZE / 2; mask > 0; mask /= 2)
#if CUDART_VERSION >= 9000
        val += __shfl_xor_sync(0xffffffff, val, mask);
#else
        val += __shfl_xor(val, mask);
#endif
    return val;
}

__global__ void backward_scale_channels_kernel(float *in_w_h_c_delta, int size, int channel_size,
    float *in_scales_c, float *out_from_delta,
    float *in_from_output, float *out_state_delta)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    int osd_index = index / channel_size;

    if (index < size) {
        //out_state_delta[osd_index] += in_w_h_c_delta[index] * in_from_output[index]; // l.delta * from  (should be divided by channel_size?)

        int warp_id = index / 32;
        int index_warp_start = warp_id * 32;
        int osd_index_warp_start = index_warp_start / channel_size;
        int osd_index_warp_end = (index_warp_start + 31) / channel_size;

        if (osd_index_warp_start == osd_index_warp_end) // all thread in warp process the same channel
        {
            float sum = warpAllReduceSum(in_w_h_c_delta[index] * in_from_output[index]); // l.delta * from
            if (threadIdx.x % 32 == 0) {
                atomicAdd(&out_state_delta[osd_index], sum);
                //out_state_delta[osd_index] += sum;
            }
        }
        else {
            atomicAdd(&out_state_delta[osd_index], in_w_h_c_delta[index] * in_from_output[index]); // l.delta * from
        }

        out_from_delta[index] += in_scales_c[osd_index] * in_w_h_c_delta[index]; // input * l.delta  // atomic isn't required here
    }
}

extern "C" void backward_scale_channels_gpu(float *in_w_h_c_delta, int size, int channel_size,
    float *in_scales_c, float *out_from_delta,
    float *in_from_output, float *out_state_delta)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    backward_scale_channels_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> > (in_w_h_c_delta, size, channel_size,
        in_scales_c, out_from_delta,
        in_from_output, out_state_delta);

    CHECK_CUDA(hipPeekAtLastError());
}
